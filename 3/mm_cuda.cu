#define N 10000
#define NR N
#define NC N

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "hip/hip_runtime.h"

void printMat(float A[NR][NC]);
void initMat(float A[NR][NC], float B[NR][NC]);

__global__ void multiply(float *A, float *B, float *C);

int main(){
	static float A[NR][NC];

	static float B[NR][NC];

	static float C[NR][NC] = {{0}}; /* initialize to 0 */

	clock_t start_time, end_time;
	double elapsed;

	float *dev_A, *dev_B, *dev_C;
	int size = NR*NC*sizeof(float);

	start_time = clock();

	hipMalloc((void **)&dev_A,size);
	hipMalloc((void **)&dev_B,size);
	hipMalloc((void **)&dev_C,size);
	
	initMat(A,B);		/* fills A with random floats */
	
	hipMemcpy(dev_A,&A,size,hipMemcpyHostToDevice);	
	hipMemcpy(dev_B,&B,size,hipMemcpyHostToDevice);	
//	cudaMemcpy(dev_C,&C,size,cudaMemcpyHostToDevice);	

	/* decide block sizes
	   call the function
		-init dev_C to 0

	   */
	hipMemcpy(&C,dev_C,size,hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
	end_time = clock();
	elapsed = ( (double) (end_time-start_time))/ CLOCKS_PER_SEC;
	
	// printMat(C);
	
	printf(" \n Time taken is %f \n",elapsed);
	
	return 0;
}

void printMat(float A[NR][NC]){

	int i,j;

	for( i=0; i<NR; i++ ){
		printf("ROW %d:",i+1);
		for( j=0; j<NC; j++ ){
			printf("%.3f\t",A[i][j]);	
		}
		printf("\n");
	}

}

void initMat(float A[NR][NC],float B[NR][NC]){

	int i,j;

	for( i=0; i < NR; i++){
		for( j=0; j<NC; j++){
			A[i][j] = i+j;
			B[i][j] = i*j;
		}
	} 

}
__global__ void multiply(float *A, float *B, float *C){


}
