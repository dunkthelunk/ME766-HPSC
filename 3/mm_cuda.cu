#define N 64
#define NR N
#define NC N

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "hip/hip_runtime.h"

void printMat(float A[NR][NC]);
void initMat(float A[NR][NC], float B[NR][NC]);

__global__ void multiply(float *A, float *B, float *C);

int main(){
	static float A[NR][NC];

	static float B[NR][NC];

	static float C[NR][NC] = {{0}}; /* initialize to 0 */

	clock_t start_time, end_time;
	double elapsed;

	float *dev_A, *dev_B, *dev_C;
	int size = NR*NC*sizeof(float);

	start_time = clock();

	hipMalloc((void **)&dev_A,size);
	hipMalloc((void **)&dev_B,size);
	hipMalloc((void **)&dev_C,size);
	
	initMat(A,B);		/* fills A with random floats */
	
	hipMemcpy(dev_A,&A,size,hipMemcpyHostToDevice);	
	hipMemcpy(dev_B,&B,size,hipMemcpyHostToDevice);	

	dim3 dimGrid(N/32,N/32);
	dim3 dimBlock(32,32);
	multiply<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_C);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy(&C,dev_C,size,hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	end_time = clock();
	elapsed = ( (double) (end_time-start_time))/ CLOCKS_PER_SEC;

	printMat(C);

	printf(" \n Time taken is %f \n",elapsed);

	return 0;
}

void printMat(float A[NR][NC]){

	int i,j;

	for( i=0; i<NR; i++ ){
		printf("ROW %d:",i+1);
		for( j=0; j<NC; j++ ){
			printf("%.3f\t",A[i][j]);	
		}
		printf("\n");
	}

}

void initMat(float A[NR][NC],float B[NR][NC]){

	int i,j;

	for( i=0; i < NR; i++){
		for( j=0; j<NC; j++){
			A[i][j] = i+j;
			B[i][j] = i*j;
		}
	} 

}
__global__ void multiply(float *A, float *B, float *C){

	// get block position in grid
	// int blockRow = blockIdx.y;
	// int blockCol = blockIdx.x;

	// get thread position in block
	int row = threadIdx.y;
	int col = threadIdx.x;

	// get absolute position
	int absRow = blockIdx.y*blockDim.y + threadIdx.y;
	int absCol = blockIdx.x*blockDim.x + threadIdx.x;
	int index = absRow*NC + absCol; // location in contiguous 1-d

	int j;
	int sum = 0;
	for(j=0;j<NC/32;j++){
		__shared__ float Apatch[32][32];
		__shared__ float Bpatch[32][32];

		//fetch the corresponding rows and cols of A,B; each thread gets one element
		Apatch[row][col] = A[absRow*NC+j*32+col];
		Bpatch[row][col] = B[absCol+j*32*NC+row*NC];
		__syncthreads();

		int i;
		for(i=0; i<32; i++) sum += Apatch[row][i]*Bpatch[i][col];
		__syncthreads();
	}
	
	C[index] = sum;

}
