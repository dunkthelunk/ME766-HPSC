#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
	hipDeviceProp_t Props;
	hipGetDeviceProperties(&Props,0);

	printf("shared mem: %d\n",Props.sharedMemPerBlock);
	printf("max threads per block : %d\n",Props.maxThreadsPerBlock);
	printf("max blocks: %d\n",Props.maxGridSize[0]);
	printf("total Const mem: %d\n",Props.totalConstMem);

}
