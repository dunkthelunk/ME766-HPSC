#define N 1024
#define NR N
#define NC N

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "hip/hip_runtime.h"

void printMat(float A[NR][NC]);
void initMat(float A[NR][NC], float B[NR][NC]);

__global__ void matSum(float *A, float *B, float *C);

int main(){
	static float A[NR][NC];

	static float B[NR][NC];

	static float C[NR][NC] = {{0}}; /* initialize to 0 */

	clock_t start_time, end_time;
	double elapsed;

	float *dev_A, *dev_B, *dev_C;
	int size = NR*NC*sizeof(float);

	start_time = clock();

	hipMalloc((void **)&dev_A,size);
	hipMalloc((void **)&dev_B,size);
	hipMalloc((void **)&dev_C,size);
	
	initMat(A,B);		/* fills A with random floats */
	
	hipMemcpy(dev_A,&A,size,hipMemcpyHostToDevice);	
	hipMemcpy(dev_B,&B,size,hipMemcpyHostToDevice);	
//	cudaMemcpy(dev_C,&C,size,cudaMemcpyHostToDevice);	

	dim3 dimGrid(N/32,N/32);
	dim3 dimBlock(32,32,1);	

	matSum<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_C);
	
	hipMemcpy(&C,dev_C,size,hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
	end_time = clock();
	elapsed = ( (double) (end_time-start_time))/ CLOCKS_PER_SEC;
	
	printMat(C);
	
	printf(" \n Time taken is %f \n",elapsed);
	
	return 0;
}

void printMat(float A[NR][NC]){

	int i,j;

	for( i=0; i<NR; i++ ){
		printf("ROW %d:",i+1);
		for( j=0; j<NC; j++ ){
			printf("%.3f\t",A[i][j]);	
		}
		printf("\n");
	}

}

void initMat(float A[NR][NC],float B[NR][NC]){

	int i,j;

	for( i=0; i < NR; i++){
		for( j=0; j<NC; j++){
			A[i][j] = 1;
			B[i][j] = 1;
		}
	} 

}

__global__ void matSum(float *A, float *B, float *C){
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int index = row*NC + col;
	C[index] = A[index] + B[index];
}
