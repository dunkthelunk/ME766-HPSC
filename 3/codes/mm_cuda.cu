#define N 10016
#define NR N
#define NC N
#define BLOCKSIZE 32


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

void printMat(float A[NR][NC]);
void initMat(float A[NR][NC], float B[NR][NC]);
__global__ void multiply(float *A, float *B, float *C);

int main(){
	static float A[NR][NC];

	static float B[NR][NC];

	static float C[NR][NC] = {{0}}; /* initialize to 0 */

	clock_t start_time, end_time;
	double elapsed;

	float *dev_A, *dev_B, *dev_C;
	int size = NR*NC*sizeof(float);

	start_time = clock();

	hipMalloc((void **)&dev_A,size);
	hipMalloc((void **)&dev_B,size);
	hipMalloc((void **)&dev_C,size);
	
	initMat(A,B);		
	
	hipMemcpy(dev_A,&A,size,hipMemcpyHostToDevice);	
	hipMemcpy(dev_B,&B,size,hipMemcpyHostToDevice);	

	dim3 dimGrid(N/BLOCKSIZE,N/BLOCKSIZE);
	dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);

	multiply<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_C);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	hipMemcpy(&C,dev_C,size,hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	end_time = clock();
	elapsed = ( (double) (end_time-start_time))/ CLOCKS_PER_SEC;

	//printMat(C);

	printf(" \n Time taken is %f \n",elapsed);

	return 0;
}

void printMat(float A[NR][NC]){

	int i,j;

	for( i=0; i<NR; i++ ){
		printf("ROW %d:",i+1);
		for( j=0; j<NC; j++ ){
			printf("%.3f\t",A[i][j]);	
		}
		printf("\n");
	}

}

void initMat(float A[NR][NC],float B[NR][NC]){

	int i,j;

	for( i=0; i < NR; i++){
		for( j=0; j<NC; j++){
			A[i][j] = i+j;
			B[i][j] = i*j;
		}
	} 

}
__global__ void multiply(float *A, float *B, float *C){

	// thread position in block
	int row = threadIdx.y;
	int col = threadIdx.x;

	// absolute position
	int absRow = blockIdx.y*blockDim.y + threadIdx.y;
	int absCol = blockIdx.x*blockDim.x + threadIdx.x;
	int index = absRow*NC + absCol; // location in contiguous 1-d

	int j;
	int sum = 0;
	for(j=0;j<NC/BLOCKSIZE;j++){
		__shared__ float Apatch[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bpatch[BLOCKSIZE][BLOCKSIZE];

		// fetch the corresponding rows and cols of A,B
		// each thread gets one element
		Apatch[row][col] = A[absRow*NC+j*BLOCKSIZE+col];
		Bpatch[row][col] = B[absCol+j*BLOCKSIZE*NC+row*NC];
		__syncthreads();

		int i;
		for(i=0; i<BLOCKSIZE; i++) sum += Apatch[row][i]*Bpatch[i][col];
		__syncthreads();
	}

	C[index] = sum;

}
